
#include <hip/hip_runtime.h>
#include <iostream>
#include <filesystem>
#include <cstdlib>

using namespace std;

int main(int argc, char** argv) {
    
    filesystem::path filePath(argv[1]);
    string extension = argv[2];

    string command = "ffmpeg -i " + filePath.generic_string() + " " + filePath.filename().stem().string() + "." + extension;
    system(command.c_str());

    return 0;
}
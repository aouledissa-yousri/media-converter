
#include <hip/hip_runtime.h>
#include <iostream>
#include <filesystem>
#include <cstdlib>

using namespace std;

int main(int argc, char** argv) {
    
    filesystem::path filePath(argv[1]);
    string extension = argv[2];

    string command = "ffmpeg -hwaccel cuda -hwaccel_output_format cude -i " + filePath.generic_string() + " " + filePath.filename().stem().string() + "." + extension;
    system(command.c_str());

    return 0;
}

#include <hip/hip_runtime.h>
#include <iostream>
#include <filesystem>
#include <cstdlib>

using namespace std;

int main(int argc, char** argv) {
    
    filesystem::path filePath(argv[1]);

    string command = "ffmpeg -hwaccel cuda -hwaccel_output_format cude -i " + filePath.generic_string() + " " + filePath.filename().stem().string() + ".mp3";
    system(command.c_str());

    return 0;
}